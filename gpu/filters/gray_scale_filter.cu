#include "hip/hip_runtime.h"
#include <iostream>
#include "gray_scale_filter.cuh"

#define GRID_SIZE 1024
#define BLOCK_SIZE 1024

#define CHANNELS_NUM 3

/**
 * Converts a color image to grayscale using CUDA kernel parallelization.
 *
 * The input image is assumed to have 3 color channels (red, green, and blue) per pixel,
 * and the output grayscale image is stored in a separate buffer allocated by the caller.
 *
 * @param image A pointer to the input image data.
 * @param gray_scaled_image A pointer to a buffer that will hold the resulting grayscale image data.
 * @param width The width of the image in pixels.
 * @param height The height of the image in pixels.
 */
__global__
static void convert_to_gray_scale_kernel(const ubyte *image, ubyte *gray_scaled_image, size_t width, size_t height) {
    // Calculate the index of the current pixel in the image
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    // Only execute the kernel for valid pixel indices
    if (i < width * height) {
        // Calculate the index of the current pixel in the input image
        size_t image_index = i * CHANNELS_NUM;

        // Convert the pixel to grayscale using the specified weights for each channel
        gray_scaled_image[i] = (ubyte) (0.21 * image[image_index] +
                                       0.72 * image[image_index + 1] +
                                       0.07 * image[image_index + 2]);
    }
}

/**
 * Converts a color image to grayscale.
 *
 * The input image is assumed to have 3 color channels (red, green, and blue) per pixel.
 * The output grayscale image is stored in a separate buffer allocated by this function.
 *
 * @param image A pointer to the input image data.
 * @param gray_scaled_image A pointer to a buffer that will hold the resulting grayscale image data.
 * @param width The width of the image in pixels.
 * @param height The height of the image in pixels.
 * @param channels The number of color channels per pixel in the input image (should be 3 for RGB images).
 * @return 0 if the conversion succeeded, or 1 if memory allocation failed.
 */
int convert_to_gray_scale(const ubyte *image, ubyte **gray_scaled_image, size_t width, size_t height,
                          size_t channels) {


    // Check if the input image and channels are valid
    if (image == nullptr || channels != 3) {
        std::cout << "Invalid input image or number of channels. Expected a 3-channel RGB image.\n";
        return 1;
    }


    // allocate memory for the gray scaled image
    *gray_scaled_image = (ubyte *) malloc(width * height * sizeof(ubyte));

    // allocate memory for the image in the device
    ubyte *d_image;
    hipMalloc((void **) &d_image, width * height * channels * sizeof(ubyte));

    // allocate memory for the gray scaled image in the device
    ubyte *d_gray_scaled_image;
    hipMalloc((void **) &d_gray_scaled_image, width * height * sizeof(ubyte));

    // copy the image to the device
    hipMemcpy(d_image, image, width * height * channels * sizeof(ubyte), hipMemcpyHostToDevice);

    // calculate the number of blocks and threads
    size_t number_of_blocks = width * height / BLOCK_SIZE;
    if ((width * height) % BLOCK_SIZE != 0) {
        number_of_blocks++;
    }

    if (number_of_blocks > GRID_SIZE) {
        number_of_blocks = GRID_SIZE;
    }

    // call the kernel
    convert_to_gray_scale_kernel<<<number_of_blocks, BLOCK_SIZE>>>
            (
                    d_image, d_gray_scaled_image,
                    width, height
            );

    // copy the gray scaled image to the host
    hipMemcpy(*gray_scaled_image, d_gray_scaled_image, width * height * sizeof(ubyte),
               hipMemcpyDeviceToHost);

    // free the memory
    hipFree(d_image);
    hipFree(d_gray_scaled_image);


    return 0;
}
