#include "hip/hip_runtime.h"
#include <iostream>
#include "brightness_filter.cuh"

__global__
void brightness_change_kernel(const ubyte *gray_scaled_img, ubyte *brightness_changed_img, size_t width, size_t height,
                              size_t channels,
                              int brightness_change) {
    // get the thread id
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // check if the thread is in the image
    if (x < width && y < height) {
        // get the index of the pixel
        size_t index = y * width + x;

        // change the brightness
        int val = gray_scaled_img[index] + brightness_change;
        val = val > UCHAR_MAX ? UCHAR_MAX : val < 0 ? 0 : val;

        // set the pixel
        brightness_changed_img[index] = val;
    }
}


int
change_brightness(const ubyte *gray_scaled_img, ubyte **brightness_changed_img, size_t width, size_t height,
                  size_t channels,
                  int brightness_change) {

    // Check if the input image and channels are valid
    if (gray_scaled_img == nullptr || channels != 1) {
        std::cout << "Invalid input image or number of channels. Expected a single-channel grayscale image.\n";
        return 1;
    }

    // allocate memory for the gray scale image
    *brightness_changed_img = (ubyte *) malloc(width * height * sizeof(ubyte));

    // check if the memory was allocated
    if (*brightness_changed_img == nullptr) {
        std::cout << "Failed to allocate memory for the gray scaled image!\n";
        return 1;
    }

    // allocate memory for the device image
    ubyte *device_gray_scaled_img;
    ubyte *device_brightness_changed_img;
    hipMalloc(&device_gray_scaled_img, width * height * sizeof(ubyte));
    hipMalloc(&device_brightness_changed_img, width * height * sizeof(ubyte));

    // copy the image to the device
    hipMemcpy(device_gray_scaled_img, gray_scaled_img, width * height * sizeof(ubyte),
               hipMemcpyHostToDevice);

    // calculate the number of blocks and threads
    dim3 block_size(32, 32);
    dim3 grid_size(width / block_size.x, height / block_size.y);
    if (width % block_size.x != 0) {
        grid_size.x++;
    }
    if (height % block_size.y != 0) {
        grid_size.y++;
    }

    // call the kernel
    brightness_change_kernel<<<grid_size, block_size>>>(device_gray_scaled_img,
                                                        device_brightness_changed_img, width,
                                                        height, channels, brightness_change);

    // copy the image from the device
    hipMemcpy(*brightness_changed_img, device_brightness_changed_img, width * height * sizeof(ubyte),
               hipMemcpyDeviceToHost);

    // free the memory
    hipFree(device_gray_scaled_img);
    hipFree(device_brightness_changed_img);

    return 0;
}
