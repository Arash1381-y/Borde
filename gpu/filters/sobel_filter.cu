#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include "sobel_filter.cuh"


#define KERNEL_WIDTH 3
#define KERNEL_HEIGHT 3


__device__
static ubyte strength_edge(long long value,
                           ubyte threshold,
                           double effect_ratio);

__device__
static ubyte clip_to_ubyte(long long val);

__device__
static void extract_kernel(const ubyte *input,
                           ubyte *output,
                           size_t height, size_t width,
                           size_t kernel_height, size_t kernel_width,
                           size_t i_index, size_t j_index);

/**
 * Applies a 2D convolution to an input image using
 *
 * @param n_row number of kernel rows
 * @param n_col number of kernel cols
 * @param input input array
 * @param kernel kernel array
 * @param output result of convolution
 */
__device__
static void convolve_2d(size_t n_row, size_t n_col,
                        const ubyte *input,
                        const byte *kernel,
                        ubyte *output) {
    long long temp = 0;

    // error checks
    assert(n_row * n_col < ULONG_LONG_MAX && "The kernel size is not supported !");

    for (size_t i = 0; i < n_row * n_col; i++) {
        temp += kernel[i] * input[i];
    }

    *output = clip_to_ubyte(temp);
}


__global__
void gpu_operate_2d_conv(ubyte *image,
                         ubyte *out_image,
                         size_t height, size_t width,
                         byte *kernel,
                         ubyte threshold) {

    size_t i_index = blockIdx.y * blockDim.y + threadIdx.y;
    size_t j_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (i_index < height && j_index < width) {
        ubyte img_sec[KERNEL_HEIGHT * KERNEL_WIDTH]; // section of image (separate by channel)

        extract_kernel(image,
                       img_sec,
                       height, width,
                       KERNEL_HEIGHT, KERNEL_WIDTH,
                       i_index, j_index);

        convolve_2d(KERNEL_HEIGHT, KERNEL_WIDTH,
                    img_sec, kernel,
                    &out_image[i_index * width + j_index]);
    }
}

/**
 * CUDA kernel that applies the Sobel filter to an input image to detect edges.
 *
 * @param image: Pointer to the input image data stored as an array of unsigned bytes (`ubyte`).
 * @param out_image: Pointer to the output image data that will contain the detected edges.
 * @param height: The height of the input image, in pixels.
 * @param width: The width of the input image, in pixels.
 * @param Gx: Pointer to the Sobel filter kernel for the horizontal direction.
 * @param Gy: Pointer to the Sobel filter kernel for the vertical direction.
 * @param apply_threshold: A boolean flag indicating whether to apply the threshold to the edge strength values.
 * @param threshold: A threshold value used to filter out weak edges.
 * @param strength_ratio: A ratio used to adjust the strength of the edge detection.
 */
__global__
static void detect_edges_sobel(const ubyte *image,
                               ubyte *out_image,
                               size_t height, size_t width,
                               const byte *Gx, const byte *Gy,
                               bool apply_threshold,
                               ubyte threshold,
                               double strength_ratio) {

    size_t row_index = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (row_index < height && col_index < width) {
        // Extract the kernel centered at the current pixel
        ubyte kernel_sec[KERNEL_HEIGHT * KERNEL_WIDTH], x_c, y_c;
        extract_kernel(image,
                       kernel_sec,
                       height, width,
                       KERNEL_HEIGHT, KERNEL_WIDTH,
                       row_index, col_index);

        // Convolve the kernel with the Sobel filter kernels to calculate the horizontal and vertical gradients
        convolve_2d(KERNEL_HEIGHT, KERNEL_WIDTH,
                    kernel_sec, Gx,
                    &x_c);

        convolve_2d(KERNEL_HEIGHT, KERNEL_WIDTH,
                    kernel_sec, Gy,
                    &y_c);

        // Calculate the edge strength using the gradient magnitudes
        auto edge_strength = (long long) hypotf(x_c, y_c);

        // Apply the threshold to filter out weak edges if required
        if (apply_threshold) {
            out_image[row_index * width + col_index] = strength_edge(edge_strength, threshold, strength_ratio);
        } else {
            out_image[row_index * width + col_index] = clip_to_ubyte(edge_strength);
        }
    }
}

/**
 * Detect edges in an input image using the Sobel filter.
 *
 * @param image: A pointer to the input image data, stored as an array of unsigned bytes (`ubyte`).
 * @param edges_detected_image: A pointer to a pointer that will be used to store the output image data, which will
 * contain the detected edges.
 * @param width: The width of the input image, in pixels.
 * @param height: The height of the input image, in pixels.
 * @param threshold: A threshold value used to filter out weak edges.
 * @param strength_ratio: A ratio used to adjust the strength of the edge detection.
 * @param dir: A flag that determines the direction of the edge detection. If `dir` is 0, the function will detect edges
 * in the horizontal direction, if it is 1, it will detect edges in the vertical direction, and if it is any other value,
 * it will detect edges in both directions.
 *
 * @return: Returns 0 if the function executed successfully, and 1 if there was an error (such as an invalid input image
 * or failure to allocate memory for the output image).
 */
int detect_edges(const ubyte *image,
                 ubyte **edges_detected_image,
                 size_t width, size_t height,
                 ubyte threshold,
                 double strength_ratio,
                 short dir) {

    // Check if the input image is valid
    if (image == nullptr) {
        std::cout << "Invalid input image\n";
        return 1;
    }

    // Initialize the output image
    *edges_detected_image = (ubyte *) malloc(width * height * sizeof(ubyte));
    if (*edges_detected_image == nullptr) return 1;

    // Initialize the Sobel filter kernels
    byte Gx[KERNEL_HEIGHT * KERNEL_WIDTH] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
    byte Gy[KERNEL_HEIGHT * KERNEL_WIDTH] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};

    // Initialize the device memory
    ubyte *d_image, *d_out_image;
    byte *d_Gx, *d_Gy;
    hipMalloc(&d_image, width * height * sizeof(ubyte));
    hipMalloc(&d_out_image, width * height * sizeof(ubyte));
    hipMalloc(&d_Gx, KERNEL_WIDTH * KERNEL_HEIGHT * sizeof(byte));
    hipMalloc(&d_Gy, KERNEL_WIDTH * KERNEL_HEIGHT * sizeof(byte));

    // Copy the input image and filter kernels to the device memory
    hipMemcpy(d_image, image, width * height * sizeof(ubyte), hipMemcpyHostToDevice);
    hipMemcpy(d_Gx, Gx, KERNEL_HEIGHT * KERNEL_HEIGHT * sizeof(byte), hipMemcpyHostToDevice);
    hipMemcpy(d_Gy, Gy, KERNEL_HEIGHT * KERNEL_HEIGHT * sizeof(byte), hipMemcpyHostToDevice);

    // Calculate the number of blocks and threads to use
    dim3 block_size(32, 32);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

    // Launch the CUDA kernel to detect edges
    if (dir == 0)
        gpu_operate_2d_conv<<<grid_size, block_size>>>
                (d_image,
                 d_out_image,
                 height, width,
                 d_Gx, threshold);
    else if (dir == 1)
        gpu_operate_2d_conv<<<grid_size, block_size>>>
                (d_image,
                 d_out_image,
                 height, width,
                 d_Gy, threshold);
    else
        detect_edges_sobel<<<grid_size, block_size>>>
                (d_image,
                 d_out_image,
                 height, width,
                 d_Gx, d_Gy,
                 true, threshold, strength_ratio);

    // Copy the output image back to the host memory
    hipMemcpy(*edges_detected_image, d_out_image, width * height * sizeof(ubyte), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_image);
    hipFree(d_out_image);
    hipFree(d_Gx);
    hipFree(d_Gy);
    return 0;
}


/**
 * Adjusts an input value based on a threshold and a strength ratio.
 *
 * If the input value is greater than the threshold, it is increased by a factor of
 * (1 + strength_ratio). Otherwise, it is decreased by a factor of (1 - strength_ratio).
 *
 * @param value The input value to adjust.
 * @param threshold The threshold for comparing the input value.
 * @param strength_ratio The strength ratio used to increase or decrease the input value.
 * @return The adjusted input value, clipped to the range [0, 255].
 */
__device__
static ubyte strength_edge(long long value, ubyte threshold, double strength_ratio) {
    if (value > threshold)
        return clip_to_ubyte((long long) ((double) value * (1 + strength_ratio)));
    else
        return clip_to_ubyte((long long) ((double) value * (1 - strength_ratio)));
}


/**
 * Clips a given long long value to the range of an unsigned byte (0 to 255).
 *
 * @param val The value to clip.
 * @return The clipped value, truncated to an unsigned byte in the range [0, 255].
 */
__device__
static ubyte clip_to_ubyte(long long val) {
    if (val > UCHAR_MAX || val < -1 * UCHAR_MAX) {
        return UCHAR_MAX;
    } else {
        return abs((int) val);
    }
}

/**
 * Extracts a sub-matrix from an input matrix centered at the given i and j indices.
 *
 * The sub-matrix is extracted using a kernel of the given height and width, and
 * the resulting sub-matrix is stored in the output array.
 *
 * If any part of the kernel extends beyond the bounds of the input matrix, the
 * corresponding elements of the output array are set to zero.
 *
 * @param input A pointer to the input matrix data.
 * @param output A pointer to the output sub-matrix data.
 * @param height The number of rows in the input matrix.
 * @param width The number of columns in the input matrix.
 * @param kernel_height The height of the kernel used to extract the sub-matrix.
 * @param kernel_width The width of the kernel used to extract the sub-matrix.
 * @param i_index The row index at the center of the sub-matrix.
 * @param j_index The column index at the center of the sub-matrix.
 */
__device__
static void extract_kernel(const ubyte *input,
                           ubyte *output,
                           size_t height, size_t width,
                           size_t kernel_height, size_t kernel_width,
                           size_t i_index, size_t j_index) {

    long i_dist, j_dist, dist_ij;
    dist_ij = ((long) kernel_width - 1) / 2;

    for (int i = 0; i < kernel_width; i++) {
        i_dist = (long) i_index + i - dist_ij;
        for (int j = 0; j < kernel_height; j++) {
            j_dist = (long) j_index + j - dist_ij;

            // Check if the current index is out of bounds
            if (i_dist >= height || i_dist < 0 || j_dist >= width || j_dist < 0) {
                output[i * kernel_width + j] = 0;
            } else {
                // Compute the index of the current element in the input matrix
                size_t index = i_dist * width + j_dist;
                output[i * kernel_width + j] = input[index];
            }
        }
    }
}